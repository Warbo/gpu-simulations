#include "hip/hip_runtime.h"
/*
 * CUDA code for making particles interact.
 * In this version, the kernel function is one big function
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "linkedlists.c"

__global__ void do_cell(particle* all_particles, int cell_size) {
	// This begins calculation of the particle interactions

	// Each block needs access to its local particles and a neighbour
	// Note: We can't use 2*cell_size here since it's not constant!
	__shared__ particle local_particles[2*32];
	
	// Work out where in the array our particles start
	int offset = (gridDim.z*gridDim.y*blockIdx.x) + (gridDim.z*blockIdx.y)
			+ blockIdx.z;

	// Each thread loads its own particle to local memory
	local_particles[threadIdx.x] =
		all_particles[(cell_size * offset) + threadIdx.x];

	// Initialise the interaction values
	local_particles[threadIdx.x].x_acc = 0.0;
	
	// Now load in our neighbours and calculate interactions
	// Loop through neighbours
	for (int x_rel = -1; x_rel < 2; x_rel++) {
		for (int y_rel = -1; y_rel < 2; y_rel++) {
			for (int z_rel = -1; z_rel < 2; z_rel++) {

				// Only act if we've got a valid neighbour
				if (
					(((int)blockIdx.x)+x_rel >= 0) &&
								(((int)blockIdx.x)+x_rel < gridDim.x) &&
								(((int)blockIdx.y)+y_rel >= 0) &&
								(((int)blockIdx.y)+y_rel < gridDim.y) &&
								(((int)blockIdx.z)+z_rel >= 0) &&
								(((int)blockIdx.z)+z_rel < gridDim.z)
				   ) {

					// Load neighbouring cell to shared memory

					// This is the memory location we need to start allocating
					// from
					int offset = (gridDim.z * gridDim.y * (blockIdx.x + x_rel))
							+ (gridDim.z * (blockIdx.y + y_rel))
							+ (blockIdx.z + z_rel);

					// Each thread allocates a particle from the neighbour
					// Start at cell_size since we're filling the second half
					local_particles[cell_size + threadIdx.x] =
							all_particles[(cell_size * offset) + threadIdx.x];

					// Ensure all particles have been loaded into shared mem
					__syncthreads();

					// Work out the interactions with these neighbours
					// Calculate interactions between a particle and everything
					// in the neighbour

					// Loop through the size of the neighbour
					for (unsigned int counter = 0; counter < cell_size;
						counter++) {
						// Make our particle interact with everything in the
						// second half of local memory

						// Try doing a predictable interaction
						//particle_A->x_acc += 1;
						local_particles[threadIdx.x].x_acc += 1;
						//&(local_particles[cell_size + counter])
					}

					// Ensure everyone's finished before loading the next cell
					__syncthreads();
					
				   }
			}
		}
	}

	// Now put shared values back into global memory
	all_particles[(cell_size * offset) + threadIdx.x] =
		local_particles[threadIdx.x];

	// Make sure we're all done
	__syncthreads();
	
}

int main() {
	// Toy example for testing

	// Allocate room for a 3x3x3 grid with 32 particles each
	particle* all_particles_host = (particle*)malloc(27*32*sizeof(particle));

	// Give particles random positions
	for (int i=0; i < 27*32; i++) {
		all_particles_host[i].x = (float) (rand()/(float)(RAND_MAX));
		all_particles_host[i].y = (float) (rand()/(float)(RAND_MAX));
		all_particles_host[i].z = (float) (rand()/(float)(RAND_MAX));
	}

	// Allocate memory on the GPU
	particle* all_particles_device;
	hipMalloc((void**) &all_particles_device, 27*32*sizeof(particle));

	// Copy across our particles
	hipMemcpy(all_particles_device, all_particles_host, 27*32*sizeof(particle),
			   hipMemcpyHostToDevice);

	dim3 dimGrid(3, 3, 3);
	// Calculate the interactions
	do_cell<<<dimGrid, 32>>>(all_particles_device, 32);

	// Get results back
	hipMemcpy(all_particles_host, all_particles_device, 27*32*sizeof(particle),
hipMemcpyDeviceToHost);

	// Free up the memory
	hipFree(all_particles_device);

	for (int i=0; i<27*32; i++) {
		printf("%G\n", all_particles_host[i].x_acc);
	}

	// Exit
	return 0;
}
