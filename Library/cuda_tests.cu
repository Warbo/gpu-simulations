/*
 * Contains tests for the CUDA implementation.
 */

// Drag in what we need
#include "cuda_functions.cu"

int main() {
	/*
	 * The tests themselves.
	 */

	// How many cycles to perform
	int interaction_number = 1;
	
	// Read in our particles
	int particle_number;
	particle* p_array;
	float particle_size;
	read_particles(&p_array, &particle_number, &particle_size);
	
	// Cell maximum size
	grid the_grid;
	grid_particles(&the_grid, p_array, particle_number, particle_size);
	int cell_size = get_biggest_cell_size(&the_grid, p_array);

	// The cell size used by CUDA must be #defined at pre-processor time, so we
	// can't change it once we've got to here. If it's not big enough, give an
	// error and exit.
	if (cell_size > CELLSIZE) {
		fprintf(stderr,
			"Error: Cells need at least %i particles, only got %i\n",
			cell_size, CELLSIZE);
		fprintf(stderr, "Change #define CELLSIZE in cuda_functions.cu\n");
		return 1;
	}
	
	// Allocate room for a padded grid
	particle* all_particles_host = (particle*)malloc((unsigned int)(
		the_grid.x_size * the_grid.y_size * the_grid.z_size * CELLSIZE
	) * sizeof(particle));

	// Allocate memory on the GPU
	particle* all_particles_device;
	hipMalloc(
		(void**)&all_particles_device,
		(the_grid.x_size * the_grid.y_size * the_grid.z_size) *
			CELLSIZE * sizeof(particle)
	);

	// Copy across our particles
	hipMemcpy(all_particles_device, all_particles_host,
		(the_grid.x_size * the_grid.y_size * the_grid.z_size) *
			CELLSIZE * sizeof(particle),
		hipMemcpyHostToDevice
	);

	dim3 dimGrid(
		the_grid.x_size * the_grid.y_size * the_grid.z_size
	);

	// Run the interactions
	/*int index;
	for (index=0; index < interaction_number; index++) {
		// Calculate the interactions
		do_cell<<<dimGrid, CELLSIZE>>>(all_particles_device, CELLSIZE,
			the_grid.x_size, the_grid.y_size, the_grid.z_size);
	}*/

	// Get results back
	hipMemcpy(all_particles_host, all_particles_device,
		(the_grid.x_size * the_grid.y_size * the_grid.z_size) *
			CELLSIZE * sizeof(particle),
		hipMemcpyDeviceToHost
	);

	// Free up the memory
	hipFree(all_particles_device);

	// DEBUG
	/*for (index=0; index <
		(the_grid.x_size * the_grid.y_size * the_grid.z_size) * CELLSIZE;
	index++) {
		printf("%G\n", all_particles_host[index].x_acc);
	}*/

	// Exit
	return 0;
}
